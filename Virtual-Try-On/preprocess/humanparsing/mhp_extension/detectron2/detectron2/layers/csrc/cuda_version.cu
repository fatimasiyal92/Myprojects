// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.

#include <hip/hip_runtime_api.h>

namespace detectron2 {
int get_cudart_version() {
  return cudaRT_VERSION;
}
} // namespace detectron2
